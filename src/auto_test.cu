#include "hip/hip_runtime.h"
#include <chrono>
#include <ctime>
#include <random>
#include <thread>
#include <unordered_map>

#include "hip/hip_runtime_api.h"
#include "hash.h"
#include "utils.h"

using hash::CpuHashEntry;
using hash::CpuHashEntryBatch;
using hash::CpuHashTable;
using hash::GpuHashTable;
using hash::HybridHashEntryBatch;
using hash::HybridHashTable;

using UniformDistribution = std::uniform_int_distribution<uint>;
using Generator = std::mt19937;
using DataMap = std::unordered_map<std::string, std::string>;

static constexpr char alphanum[] = "0123456789ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz";
static constexpr uint NUM_TEST_ENTRIES = 1000000;

class HashTableTestBase {
  public:
    HashTableTestBase(std::string name) : name_{name} {
    }

    virtual ~HashTableTestBase(){};

    // Inserts every entry in the given DataMap
    virtual void insert_all(DataMap& test_data) = 0;

    // Finds every entry in the given DataMap
    virtual void find_all(DataMap& test_data, bool check_data) = 0;

    // Clears all entries in hash table
    virtual void clear() = 0;

    void test_all(DataMap& test_data, bool check_data) {
        time_t t0_insert = get_time_us();
        insert_all(test_data);
        time_t t1_insert = get_time_us();
        insert_all_times.emplace_back(t1_insert - t0_insert);
        std::cout << name_ << " Insert time: " << t1_insert - t0_insert << " us" << std::endl;

        time_t t0_find = get_time_us();
        find_all(test_data, check_data);
        time_t t1_find = get_time_us();
        find_all_times.emplace_back(t1_find - t0_find);
        std::cout << name_ << " Find time: " << t1_find - t0_find << " us" << std::endl;
    }

    void print_averages() {
        time_t insert_all_avg =
            std::accumulate(insert_all_times.begin(), insert_all_times.end(), 0) / insert_all_times.size();
        time_t find_all_avg = std::accumulate(find_all_times.begin(), find_all_times.end(), 0) / find_all_times.size();

        std::cout << name_ << " Avg Insert time: " << insert_all_avg << " us" << std::endl;
        std::cout << name_ << " Avg Find time: " << find_all_avg << " us" << std::endl;
    }

  protected:
    std::string name_;
    std::vector<time_t> insert_all_times;
    std::vector<time_t> find_all_times;
};

class CpuHashTableTest : public HashTableTestBase {
  public:
    CpuHashTableTest(std::string name) : HashTableTestBase(name) {
        h = new CpuHashTable();
        h->init();

        in_batch = new CpuHashEntryBatch;
        out_batch = new CpuHashEntryBatch;

        in_entry = new CpuHashEntry;
        out_entry = new CpuHashEntry;
    }

    ~CpuHashTableTest() {
        delete h;
        delete in_batch;
        delete out_batch;
    }

    void clear() override {
        h->clear();
    }

  protected:
    void insert_all(DataMap& test_data) override {
        for (auto& entry : test_data) {
            const std::string& key = entry.first;
            const std::string& word = entry.second;

            std::memcpy(&in_entry->key, key.c_str(), key.size());
            std::memcpy(&in_entry->word, word.c_str(), word.size());

            if (in_entry->key[0] == '\0') {
                std::cout << "found the empty key\n";
            }
            h->insert_entry(in_entry);
        }
    }

    void compare_data(DataMap& test_data, CpuHashEntry* out_entry) {
        const char* out_key = out_entry->key;
        const char* out_word = out_entry->word;

        std::string key(out_key, KEY_SIZE);
        std::string word = test_data[key];
        if (strcmp(key.c_str(), out_key) == 0 && strcmp(word.c_str(), out_word) == 0) {
            // printf("entry(%s, %s) == map_entry(%s, %s)\n", key.c_str(), word.c_str(), out_key, out_word);
        } else {
            printf("CPU: entry(%s, %s) != map_entry(%s, %s)\n", key.c_str(), word.c_str(), out_key, out_word);
            abort_with_trace();
        }
    }

    void find_all(DataMap& test_data, bool check_data) override {
        for (auto& entry : test_data) {
            const std::string& key = entry.first;

            std::memcpy(&out_entry->key, key.c_str(), key.size());
            std::memset(&out_entry->word, 0, WORD_SIZE);

            h->find_entry(out_entry);
            if (check_data) {
                compare_data(test_data, out_entry);
            }
        }
    }

    CpuHashTable* h;
    CpuHashEntryBatch* in_batch;
    CpuHashEntryBatch* out_batch;
    CpuHashEntry* in_entry;
    CpuHashEntry* out_entry;
};

class HybridHashTableTest : public HashTableTestBase {
  public:
    HybridHashTableTest(std::string name) : HashTableTestBase(name) {
        h = new HybridHashTable;
        cudaCheckErrors();

        for (uint i = 0; i < NUM_BATCHES; i++) {
            hipHostMalloc(&in_batches[i], sizeof(HybridHashEntryBatch));
        }
        hipHostMalloc(&out_batch, sizeof(HybridHashEntryBatch));
        hipDeviceSynchronize();
        cudaCheckErrors();
    }

    ~HybridHashTableTest() {
        delete h;
        for (uint i = 0; i < NUM_BATCHES; i++) {
            hipHostFree(&in_batches[i]);
        }
        hipHostFree(&out_batch);
    }

    void clear() override {
        h->clear();
    }

  protected:
    void insert_all(DataMap& test_data) override {
        uint in_batch_index = 0;
        HybridHashEntryBatch* in_batch = in_batches[in_batch_index];

        uint batch_index = 0;  // Count of entries in this batch
        for (auto& entry : test_data) {
            const std::string& key = entry.first;
            const std::string& word = entry.second;
            std::memcpy(&in_batch->keys[batch_index], key.c_str(), key.size());
            std::memcpy(&in_batch->words[batch_index], word.c_str(), word.size());
            batch_index++;

            // Insert entries using max BATCH_SIZE
            if (batch_index == BATCH_SIZE) {
                h->insert_batch(in_batch, BATCH_SIZE);
                in_batch_index++;
                in_batch = in_batches[in_batch_index];
                // cudaCheckErrors();
                batch_index = 0;
            }
        }
        // Insert remaining entries
        if (batch_index > 0 && batch_index < BATCH_SIZE) {
            h->insert_batch(in_batch, batch_index);
            // cudaCheckErrors();
        }
    }

    void compare_data(DataMap& test_data, HybridHashEntryBatch* out_batch, uint num_entries) {
        for (uint i = 0; i < num_entries; i++) {
            const char* out_key = out_batch->keys[i];
            const char* out_word = out_batch->words[i];

            std::string key(out_key, KEY_SIZE);
            auto it = test_data.find(key);
            if (it == test_data.end()) {
                printf("Key %s not found in the test data map\n", key.c_str());
                abort_with_trace();
            }
            std::string word = test_data[key];

            if (strcmp(key.c_str(), out_key) == 0 && strcmp(word.c_str(), out_word) == 0) {
                // printf("entry(%s, %s) == map_entry(%s, %s)\n", key.c_str(), word.c_str(), out_key, out_word);
            } else {
                printf("GPU: entry(%s, %s) != map_entry(%s, %s)\n", key.c_str(), word.c_str(), out_key, out_word);
                abort_with_trace();
            }
        }
    }

    void find_all(DataMap& test_data, bool check_data) override {
        uint batch_index = 0;
        for (auto& entry : test_data) {
            const std::string& key = entry.first;
            std::memcpy(&out_batch->keys[batch_index], key.c_str(), key.size());
            std::memset(&out_batch->words[batch_index], 0, WORD_SIZE);
            batch_index++;

            // Insert entries using max BATCH_SIZE
            if (batch_index == BATCH_SIZE) {
                h->find_batch(out_batch, BATCH_SIZE);

                if (check_data) {
                    cudaCheckErrors();
                    compare_data(test_data, out_batch, BATCH_SIZE);
                }
                batch_index = 0;
            }
        }
        // Check remaining entries
        if (batch_index > 0 && batch_index < BATCH_SIZE) {
            h->find_batch(out_batch, batch_index);

            if (check_data) {
                cudaCheckErrors();
                compare_data(test_data, out_batch, batch_index);
            }
        }
    }

  protected:
    static constexpr uint NUM_BATCHES = (NUM_TEST_ENTRIES / BATCH_SIZE) + (NUM_TEST_ENTRIES % BATCH_SIZE != 0 ? 1 : 0);

    HybridHashTable* h;
    HybridHashEntryBatch* in_batches[NUM_BATCHES];
    HybridHashEntryBatch* out_batch;
};

std::string get_random_string(UniformDistribution& char_picker, Generator& generator, uint used_space, uint length) {
    // This will make sure our string is already null terminated
    std::string s(length, 0);
    for (uint i = 0; i < used_space; i++) {
        s[i] = alphanum[char_picker(generator)];
    }
    return s;
}

int main(void) {
    Generator generator(std::chrono::system_clock::now().time_since_epoch().count());

    UniformDistribution char_picker(0, sizeof(alphanum) - 2);  // -1 for null terminator, -1 for 0 index
    UniformDistribution key_size(1, KEY_SIZE - 1);
    UniformDistribution word_size(1, WORD_SIZE - 1);

    // Generate random key, value pairs
    DataMap test_entries;
    for (uint i = 0; i < NUM_TEST_ENTRIES; i++) {
        std::string key = get_random_string(char_picker, generator, key_size(generator), KEY_SIZE);
        std::string word = get_random_string(char_picker, generator, word_size(generator), WORD_SIZE);

        // std::cout << "Generated entry (" << key << ", " << word << ")" << std::endl;

        test_entries[key] = word;
    }

    HybridHashTableTest hybrid_tester("Hybrid");
    hipProfilerStart();
    hybrid_tester.test_all(test_entries, false);
    hipProfilerStop();

    CpuHashTableTest cpu_tester("CPU");
    cpu_tester.test_all(test_entries, false);

    return 0;
}
