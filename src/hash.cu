#include "hip/hip_runtime.h"
#include "hash.h"

__device__ void init_hash_entry(hash_entry_t* entry) {
    std::memset(entry, 0, sizeof(hash_entry_t));
    entry->occupied = false;  // redundant but just to be certain
}

__global__ void init_hash_table(int n, hash_table_t* hash_table) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        hash_entry_t entry = hash_table->entries[i];
        init_hash_entry(&entry);
    }
}

__device__ inline uint32_t fnv1a(char oneByte, uint32_t hash = SEED) {
    return (oneByte ^ hash) * PRIME;
}

__device__ uint32_t hash_function(char* key, uint32_t hash = SEED)  // fnv hash
{
    const char* ptr = (const char*)key;
    for (int i = 0; i < KEY_SIZE; i++) {
        hash = fnv1a(*ptr++, hash);
    }
    return hash % NUM_ELEMENTS;
}

__device__ bool device_memcmp(char* word1, char* word2, uint len) {
    for (uint i = 0; i < len; i++) {
        if (word1[i] != word2[i]) {
            return false;
        }
    }
    return true;
}

__device__ void HashTable::insert_entry(HashEntry* user_entry) {
    char* key = user_entry->key;
    char* word = user_entry->word;

    uint32_t hash_val = hash_function(key);
    printf("insert hash_val: %u\n", hash_val);

    HashEntryInternal* entry = &entries[hash_val];
    while (entry->occupied) {
        // If the keys match, just overwrite the data
        if (device_memcmp(key, entry->key, KEY_SIZE)) {
            break;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &entries[hash_val];
        printf("insert hash_val: %u\n", hash_val);
    }

    std::memcpy(entry->key, key, KEY_SIZE);
    std::memcpy(entry->word, word, WORD_SIZE);
    entry->occupied = true;
}

__device__ void HashTable::find_entry(HashEntry* user_entry) {
    char* key = user_entry->key;

    uint32_t hash_val = hash_function(key);
    printf("get hash_val: %u\n", hash_val);

    HashEntryInternal* entry = &entries[hash_val];
    // Loop until we reach an empty entry OR find the key
    while (entry->occupied) {
        printf("Comparing keys %s & %s\n", entry->key, key);
        if (device_memcmp(key, entry->key, KEY_SIZE)) {
            printf("Found word: %s\n", entry->word);
            std::memcpy(user_entry->word, entry->word, WORD_SIZE);
            return;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &entries[hash_val];
        printf("get hash_val: %u\n", hash_val);
    }

    // key not found, make sure the word we send back is empty
    user_entry->word[0] = 0;
    return;
}

__device__ HashEntryInternal* HashTable::get_entry(uint32_t index) {
    return &entries[index];
}

__device__ void init_hash_entry(HashEntryInternal* entry) {
    std::memset(entry, 0, sizeof(HashEntryInternal));
}

__global__ void init_hash_table(HashTable* hash_table, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint32_t i = index; i < n; i += stride) {
        HashEntryInternal* entry = hash_table->get_entry(i);
        init_hash_entry(entry);
    }
}

__global__ void hash_insert(HashTable* hash_table, HashEntry* entry) {
    hash_table->insert_entry(entry);
}

__global__ void hash_find(HashTable* hash_table, HashEntry* entry) {
    hash_table->find_entry(entry);
}

__global__ void insert_entry(hash_table_t* hash_table, char* key, char* word) {
    uint32_t hash_val = hash_function(key);
    printf("insert hash_val: %u\n", hash_val);
    hash_entry_t* entry;

    entry = &hash_table->entries[hash_val];
    while (entry->occupied) {
        // If the keys match, just overwrite the data
        if (device_memcmp(key, entry->key, KEY_SIZE)) {
            break;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &hash_table->entries[hash_val];
        printf("insert hash_val: %u\n", hash_val);
    }
    memcpy(entry->key, key, KEY_SIZE);
    memcpy(entry->word, word, WORD_SIZE);
    entry->occupied = true;
}

__global__ void get_entry(hash_table_t* hash_table, char* key, char* host_word) {
    uint32_t hash_val = hash_function(key);
    printf("get hash_val: %u\n", hash_val);
    hash_entry_t* entry;

    entry = &hash_table->entries[hash_val];
    // Loop until we reach an empty entry OR find the key
    while (entry->occupied) {
        printf("Comparing keys %s & %s\n", entry->key, key);
        if (device_memcmp(key, entry->key, KEY_SIZE)) {
            printf("Found word: %s\n", entry->word);
            memcpy(host_word, entry->word, WORD_SIZE);
            return;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &hash_table->entries[hash_val];
        printf("get hash_val: %u\n", hash_val);
    }
    host_word = nullptr;
    return;
}

void test_old_table() {
    hash_table_t* hash_table;

    hipMallocManaged(&hash_table, sizeof(hash_table_t));

    init_hash_table<<<NUM_BLOCKS, BLOCK_SIZE>>>(NUM_ELEMENTS, hash_table);
    hipDeviceSynchronize();

    char *key, *word;
    hipMallocManaged(&key, KEY_SIZE);
    hipMallocManaged(&word, WORD_SIZE);
    strcpy(key, "abcdefghijklmnopqrstuvwxyz12345");
    strcpy(word, "abcdefghijklmnopqrstuvwxyz12345abcdefghijklmnopqrstuvwxyz123456");

    insert_entry<<<1, 1>>>(hash_table, key, word);
    hipDeviceSynchronize();

    char* buffer;
    hipMallocManaged(&buffer, WORD_SIZE);
    get_entry<<<1, 1>>>(hash_table, key, buffer);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    if (memcmp(buffer, word, WORD_SIZE) == 0) {
        std::cout << "Hooray they're equal" << std::endl;
        std::cout << "Word: " << word << std::endl;
        std::cout << "Buffer: " << buffer << std::endl;
    } else {
        std::cout << "Boo I suck" << std::endl;
        std::cout << "Word: " << word << std::endl;
        std::cout << "Buffer: " << buffer << std::endl;
    }

    hipFree(hash_table);
    hipFree(key);
    hipFree(word);
    hipFree(buffer);
}

void test_new_table() {
    HashTable* h =  new HashTable;
    hipDeviceSynchronize();

    init_hash_table<<<NUM_BLOCKS, BLOCK_SIZE>>>(h, NUM_ELEMENTS);
    hipDeviceSynchronize();

    char test_key[] = "abcdefghijklmnopqrstuvwxyz12345";
    char test_word[] = "abcdefghijklmnopqrstuvwxyz12345abcdefghijklmnopqrstuvwxyz123456";

    HashEntry* in = new HashEntry(test_key, test_word);
    HashEntry* out = new HashEntry(test_key);
    hipDeviceSynchronize();

    hash_insert<<<1, 1>>>(h, in);
    hipDeviceSynchronize();
    hash_find<<<1, 1>>>(h, out);
    hipDeviceSynchronize();
    if (*in == *out) {
        std::cout << "Hooray they're equal" << std::endl;
    } else {
        std::cout << "Boo I suck" << std::endl;
    }

    delete h;
    delete in;
    delete out;
}

int main(void) {
    test_new_table();

    return 0;
}
