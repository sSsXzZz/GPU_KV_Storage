#include "hip/hip_runtime.h"
#include <thread>
#include <vector>

#include "hash.h"
#include "utils.h"

__device__ __host__ inline uint32_t fnv1a(char oneByte, uint32_t hash = SEED) {
    return (oneByte ^ hash) * PRIME;
}

__device__ __host__ uint32_t hash_function(char* key, uint32_t hash = SEED)  // fnv hash
{
    const char* ptr = (const char*)key;
    for (int i = 0; i < KEY_SIZE; i++) {
        hash = fnv1a(*ptr++, hash);
    }
    return hash % NUM_ELEMENTS;
}

// return true if words are equal for len chars
__device__ bool device_memcmp(char* word1, char* word2, uint len) {
    for (uint i = 0; i < len; i++) {
        if (word1[i] != word2[i]) {
            return false;
        }
    }
    return true;
}

namespace hash {

// ----------------------------------------------
// CPU Hash Table
// ----------------------------------------------

void CpuHashTable::init() {
    std::memset(entries, 0, sizeof(HashEntryInternal) * NUM_ELEMENTS);
}

void CpuHashTable::insert_entry(CpuHashEntry* user_entry) {
    char* key = user_entry->key;
    char* word = user_entry->word;

    uint32_t hash_val = hash_function(key);
    // printf("insert hash_val: %u\n", hash_val);
    // printf("Request to insert element with key %s, word %s\n", user_entry->key, user_entry->word);

    HashEntryInternal* entry = &entries[hash_val];
    // TODO handle case when hash table is full
    while (entry->occupied) {
        // If the keys match, just overwrite the data
        if (std::memcmp(key, entry->key, KEY_SIZE) == 0) {
            break;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &entries[hash_val];
        // printf("insert hash_val: %u\n", hash_val);
    }

    std::memcpy(entry->key, key, KEY_SIZE);
    std::memcpy(entry->word, word, WORD_SIZE);
    entry->occupied = true;
    // printf("Inserted element at hash_val %u, key %s, word %s\n", hash_val, entry->key, entry->word);
}

void CpuHashTable::find_entry(CpuHashEntry* user_entry) {
    char* key = user_entry->key;

    uint32_t hash_val = hash_function(key);
    // printf("get hash_val: %u\n", hash_val);

    HashEntryInternal* entry = &entries[hash_val];
    // Loop until we reach an empty entry OR find the key
    while (entry->occupied) {
        // //printf("Comparing keys %s & %s\n", entry->key, key);
        if (std::memcmp(key, entry->key, KEY_SIZE) == 0) {
            // printf("Found word: %s\n", entry->word);
            std::memcpy(user_entry->word, entry->word, WORD_SIZE);
            return;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &entries[hash_val];
        // printf("get hash_val: %u\n", hash_val);
    }

    // key not found, make sure the word we send back is empty
    user_entry->word[0] = 0;
    return;
}

void CpuHashTable::clear() {
    std::memset(entries, 0, sizeof(HashEntryInternal) * NUM_ELEMENTS);
}

void CpuHashTable::debug_print_entries() {
    printf("_____ ALL CPU ENTRIES _____\n");
    for (uint32_t i = 0; i < NUM_ELEMENTS; i++) {
        HashEntryInternal* entry = &entries[i];
        if (entry->occupied) {
            printf("%u: (%s, %s)\n", i, entry->key, entry->word);
        }
    }
    printf("___________________________\n");
}

// ----------------------------------------------
// Hybrid/GPU Hash Table
// ----------------------------------------------

__global__ void gpu_init(GpuHashTable* hash_table) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint32_t i = index; i < NUM_ELEMENTS; i += stride) {
        std::memset(&hash_table->entries[i], 0, sizeof(HashEntryInternal));
    }
}

__global__ void gpu_insert_batch(GpuHashTable* hash_table, HybridInsertBatch* insert_batch, uint num_entries) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint32_t i = index; i < num_entries; i += stride) {
        char* key = insert_batch->keys[i];
        char* word = insert_batch->words[i];
        uint32_t location = insert_batch->locations[i];

        hash_table->entries[location].occupied = true;
        std::memcpy(&hash_table->entries[location].key, key, KEY_SIZE);
        std::memcpy(&hash_table->entries[location].word, word, WORD_SIZE);
    }
}

__global__ void gpu_find_batch(GpuHashTable* hash_table, HybridFindBatchInput* input_find_batch,
                               HybridFindBatchOutput* output_find_batch, uint num_entries) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint32_t i = index; i < num_entries; i += stride) {
        // find location (location) that memory is stored
        char* key = input_find_batch->keys[i];
        uint32_t location = hash_function(key);
        HashEntryInternal* entry = &hash_table->entries[location];
        while (entry->occupied) {
            // IF a match is found, copy the word and move onto next batch entry
            if (device_memcmp(key, entry->key, KEY_SIZE) == true) {
                std::memcpy(&output_find_batch->words[i], &entry->word, WORD_SIZE);
                output_find_batch->entry_found[i] = true;
                break;
            }
            location = (location + 1) % NUM_ELEMENTS;
            entry = &hash_table->entries[location];
        }

        // Do nothing if the entry was not found
    }
}

HybridHashTable::HybridHashTable() : stream_count{0} {
    word_storage = new GpuHashTable;
    hipDeviceSynchronize();
    cudaCheckErrors();

    std::memset(key_storage, 0, sizeof(HybridHashEntryInternal) * NUM_ELEMENTS);

    gpu_init<<<NUM_BLOCKS_ALL, BLOCK_SIZE>>>(word_storage);
    hipDeviceSynchronize();
    cudaCheckErrors();

    hipMalloc(&insert_buf, sizeof(HybridInsertBatch));
    for (uint i = 0; i < MAX_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
        hipMalloc(&input_find_bufs[i], sizeof(HybridFindBatchInput));
        hipMalloc(&output_find_bufs[i], sizeof(HybridFindBatchOutput));
    }
}

uint32_t HybridHashTable::find_location(char key[KEY_SIZE]) {
    static time_t thash = 0;
    static time_t tfind = 0;
    time_t t0;

    t0 = get_time_us();
    uint32_t hash_val = hash_function(key);
    thash += get_time_us() - t0;

    t0 = get_time_us();
    HybridHashEntryInternal* entry = &key_storage[hash_val];
    // TODO handle case when hash table is full
    while (entry->occupied) {
        // If the keys match, just overwrite the data
        if (std::memcmp(key, entry->key, KEY_SIZE) == 0) {
            break;
        }
        hash_val = (hash_val + 1) % NUM_ELEMENTS;
        entry = &key_storage[hash_val];
    }
    tfind += get_time_us() - t0;

    return hash_val;
}

void HybridHashTable::insert_batch(HybridInsertBatch* insert_batch, uint num_entries) {
    // iterate through entries, find the key location, and populate the internal_batch with the locations
    time_t t0 = get_time_us();
    for (uint i = 0; i < num_entries; i++) {
        uint32_t location = find_location(insert_batch->keys[i]);
        insert_batch->locations[i] = location;

        // Set key storage to occupied and copy key
        key_storage[location].occupied = true;
        std::memcpy(key_storage[location].key, insert_batch->keys[i], KEY_SIZE);
    }
    // printf("insert_batch - %lu us to find locations\n", get_time_us() - t0);

    // MemcpyAsync on default stream so that we don't block on this call
    hipMemcpyAsync(insert_buf, insert_batch, sizeof(HybridInsertBatch), hipMemcpyHostToDevice, DEFAULT_STREAM);
    gpu_insert_batch<<<NUM_BLOCKS_BATCH, BLOCK_SIZE>>>(word_storage, insert_buf, num_entries);
}

void HybridHashTable::sync_inserts() {
    hipStreamSynchronize(DEFAULT_STREAM);
}

void HybridHashTable::find_batch(HybridFindBatchInput* input_find_batch, HybridFindBatchOutput* output_find_batch,
                                 uint num_entries) {
    uint stream = 0;

    {
        std::lock_guard<std::mutex> lg(find_lock);
        stream = stream_count;
        stream_count = (stream_count + 1) % MAX_STREAMS;

        hipMemcpyAsync(input_find_bufs[stream], input_find_batch, sizeof(HybridFindBatchInput), hipMemcpyHostToDevice,
                        streams[stream]);
        gpu_find_batch<<<NUM_BLOCKS_BATCH, BLOCK_SIZE, 0, streams[stream]>>>(word_storage, input_find_bufs[stream],
                                                                             output_find_bufs[stream], num_entries);
        hipMemcpyAsync(output_find_batch, output_find_bufs[stream], sizeof(HybridFindBatchOutput),
                        hipMemcpyDeviceToHost, streams[stream]);
    }

    hipStreamSynchronize(streams[stream]);
}

void HybridHashTable::clear() {
    std::memset(key_storage, 0, sizeof(HybridHashEntryInternal) * NUM_ELEMENTS);
    hipMemset(word_storage, 0, sizeof(GpuHashTable));
    cudaCheckErrors();
    hipDeviceSynchronize();
}

__global__ void debug_print_entries_internal(GpuHashTable* hash_table) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (uint32_t i = index; i < NUM_ELEMENTS; i += stride) {
        HashEntryInternal* entry = &hash_table->entries[i];
        if (entry->occupied) {
            printf("%u: (%s, %s)\n", i, entry->key, entry->word);
        }
    }
}

void HybridHashTable::debug_print_entries() {
    debug_print_entries_internal<<<NUM_BLOCKS_ALL, BLOCK_SIZE>>>(word_storage);
}

}  // namespace hash
